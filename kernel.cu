
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>

#define PI 3.14159265
#define inf 9999.99f

__global__ void Scale_Init(float xmax, float xmin, float *pos, float *velocity, float *p_best_y, int *l_best_index, int *best_index, hiprandState *states){
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int t_index = threadIdx.x;
    
    //Rescale velocity so that it is within the bounds
    velocity[index] =(xmax - xmin) * (2.0f * velocity[index] - 1.0f);
    
    //Rescale pos so that it is within the bounds
    pos[index] = xmax * (2.0f * pos[index] - 1.0f);
    
    //Initializing p_best_y to infinity and local best to self
    if (t_index == 0)
    {
        p_best_y[blockIdx.x] = inf;
        l_best_index[blockIdx.x] = blockIdx.x;
        best_index[blockIdx.x] = blockIdx.x;
    }

    //call of Curand_init on a specific curandState, seed and no offset for each thread
    hiprand_init(index, index, 0, &states[index]);
}

// Kernel to compute the actual iterations of the updates 
__global__ void Iterations(float xmax, float xmin, float *pos, float *velocity, float *p_best_pos,float *p_best_y, int *l_best_index, int *best_index, hiprandState *states, float c_1, float c_2, float inertia, float vmax, float chi, int N, int D){
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    
    float r1, r2;
    
    float personal_best;
    int local_best;
    hiprandState local_state = states[index];

  
    //Calculate fitness of particle
    float fitness = 0.0f;
//    float fitness2 = 1.0f;
//    float fitness1 = 0.0f;    

// For f1
    for (int i = 0; i < D; i++)
        fitness += pos[index * D + i]*pos[index * D + i];
        
// For f2
//    for (int i = 0; i < D ; i++)
//        fitness += pos[index * D + i]*pos[index * D + i] - 10*cos(2*PI*pos[index * D + i]) +10;

// For f3
//    for (int i = 0; i < D; i++)
//        fitness1 += pos[index * D + i]*pos[index * D + i];
//        fitness2 *= cos(pos[index * D + i]/sqrt((float) index * D + j)
//    fitness = (1/4000)*fitness1 - fitness2 + 1
        
// For f4
//    for (int i = 0; i < D -1; i++)
//        fitness += (100*(pos[index * D + i + 1] - pos[index * D + i]*pos[index * D + i])*(pos[index * D + i + 1] 
//    - pos[index * D + i]*pos[index * D + i]) + (pos[index * D + i] - 1)*(pos[index * D + i] - 1)) ;
        
        
  
   
    //If fitness is better, change particle best
    if (p_best_y[index] > fitness)
    {
      p_best_y[index] = fitness;
      for (int i = 0; i < D; i++)
        p_best_pos[index * D + i] = pos[index * D + i];
    }
    personal_best = p_best_y[index];
    
    //Look up for left and right neighbours
    int left = (N + index - 1) % N;
    int right = (1 + index) % N;
    
    //Set the local best index
    if (p_best_y[left] < personal_best)
      l_best_index[index] = left;
    if (p_best_y[right] < personal_best)
      l_best_index[index] = right;
    local_best = l_best_index[index];
    
    //Compute and update particle velocity and position
    for (int i = 0; i < D; i++)
    {
      int j = index * D + i;
      r1 = hiprand_uniform(&local_state);
      r2 = hiprand_uniform(&local_state);
      
      // Compute the velocity
      velocity[j] = chi * (velocity[j] + (c_1 * r1 * (p_best_pos[j] - pos[j])) + (c_1 * r2 * (p_best_pos[local_best] - pos[j])));
      
      //Ensure velocity values are within range
      if (velocity[j] > (xmax - xmin) )
        velocity[j] = (xmax - xmin);
      if (velocity[j] < -(xmax - xmin))
        velocity[j] = -(xmax - xmin);
      
      //Update the position ensuring all values are within the xmin to xmax range
      pos[j] = pos[j] + velocity[j];
      if (pos[j] > xmax)
        pos[j] = xmax;
      if (pos[j] < xmin)
        pos[j] = xmin;
    }
    
    //Set the current state of the PRNG
    states[index] = local_state;
      
}

__global__ void ReduceKernel1(float *p_best_fitness, int* best_index) {

    // Calculate global thread index based on the block and thread indices ----

    int i = threadIdx.x + blockDim.x * blockIdx.x; 
    int tx = threadIdx.x;

    __shared__ float stage[512];
    __shared__ int best[512];
    
    best[tx] = best_index[i];
    stage[tx] = p_best_fitness[i];

    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s>0; s >>=1){
    if (tx < s){
        if (stage[tx] > stage[tx + s]){
            stage[tx] = stage[tx + s];
            best[tx] = best[tx + s];
            }
        }
    __syncthreads();
    }

    if (tx == 0){
       p_best_fitness[blockIdx.x] = stage[0];
       best_index[blockIdx.x] = best[0];   
    }
    
}

__global__ void ReduceKernel2(float* p_best_pos, float *p_best_fitness, int* best_index, int D) {

    // Calculate global thread index based on the block and thread indices ----

    int i = threadIdx.x + blockDim.x * blockIdx.x; 
    int tx = threadIdx.x;
    
    __shared__ float stage[512];
    __shared__ int best[512];
    
    best[tx] = best_index[i];
    stage[tx] = p_best_fitness[i];

    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s>0; s >>=1){
    if (tx < s){
        if (stage[tx] > stage[tx + s]){
            stage[tx] = stage[tx + s];
            best[tx] = best[tx + s];
            }
        }
    __syncthreads();
    }

    if (tx == 0){
       p_best_fitness[blockIdx.x] = stage[0];
       best_index[blockIdx.x] = best[0];   
    }

    for (int j = 0; j < D; j++){
        p_best_pos[j] = p_best_pos[best[0] * D + j];}
     
   
}
