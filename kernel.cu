#include "hip/hip_runtime.h"
//Kernel to Scale velocity and position values to be between the max and min (and not between 0 and 1 anymore)
// AND initialize particle best (for each particle) + local best 

#define inf 9999.99f

__global__void Scale_Init(float *xmax, float *xmin, float *pos, float *velocity, float *p_best_y, int *l_best_index, int *best_index, hiprandState *states)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int t_index = threadIdx.x;

    //Rescale pos between xmin and xmax
    pos[index] = x_max * (2.0f * pos[index] - 1.0f);

    //Rescale velocity
    velocity[index] =(x_max - x_min) * (2.0f * velocity[index] - 1.0f);
    
    //Set PBest to infinity and LBest to self
    //Initialize array of best indices
    if (t_index == 0)
    {
        p_best_y[blockIdx.x] = inf;
        l_best_index[blockIdx.x] = blockIdx.x;
        best_index[blockIdx.x] = blockIdx.x;
    }

    //Initializing up cuRAND
    //Each thread gets a different seed, different sequence number and no offset
    hiprand_init(index, index, 0, &states[index]);
}

// Kernel to compute the actual iterations of the updates 
_global_void Iterate(float *xmax, float *xmin, float *pos, float *velocity, float *p_best_pos,float *p_best_y, int *l_best_index, int *best_index, hiprandState *states, float *c_1, float *c_2)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    
    float r1, r2;
    
    float personal_best;
    int local_best;
    hiprandState local_state = states[index];

  
    //Calculate fitness of particle
    float fitness = 0.0f;
    for (int i = 0; i < D; i++)
        fitness += pos[index * D + i] * pos[index * D +i];
  
   
    //Set PBest if fitness is better
    if (p_best_y[index] > fitness)
    {
      p_best_y[index] = fitness;
      for (int i = 0; i < D; i++)
        p_best_pos[index * D + i] = pos[index * D + i];
    }
    personal_best = p_best_y[index];
    
    //Set left and right neighbours
    int left = (N + index - 1) % N;
    int right = (1 + index) % N;
    
    //Set the local best index
    if (p_best_y[left] < personal_best)
      l_best_index[index] = left;
    if (p_best_y[right] < personal_best)
      l_best_index[index] = right;
    local_best = l_best_index[index];
    
    //Update the particle velocity and position
    for (int i = 0; i < D; i++)
    {
      int j = index * D + i;
      r1 = hiprand_uniform(&local_state);
      r2 = hiprand_uniform(&local_state);
      
      // Compute the velocity
      velocity[j] = chi * (velocity[j] + (c1 * r1 * (p_best_pos[j] - pos[j])) + (c2 * r2 * (p_best_pos[local_best] - pos[j])));
      
      //Ensure velocity values are within range
      if (velocity[j] > (xmax - xmin) )
        velocity[j] = (xmax - xmin);
      if (velocity[j] < -(xmax - xmin))
        velocity[j] = -(xmax - xmin);
      
      //Update the position ensuring all values are within the xmin to xmax range
      pos[j] = pos[j] + velocity[j];
      if (pos[j] > xmax)
        pos[j] = xmax;
      if (pos[j] < xmin)
        pos[j] = xmin;
    }
    
    //Set the current state of the PRNG
    states[index] = local_state;
      
}
