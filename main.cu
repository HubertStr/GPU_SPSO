#include "hip/hip_runtime.h"
// Let this script use GPU generated random numbers


#include <stdio.h>
#include "support.h"
#include "kernel.cu"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>


#define max_iters 2048              //number of iterations

#define inf 9999.99f                //infinity


int main(int argc, char**argv){
    Timer timer;
    unsigned int N;
    unsigned int D;
    float xmax;
    float xmin;
    float c_1;
    float c_2;
    float inertia;
    float vmax;  
    int *l_best_index, *best_index;
    float *particle_position, *particle_velocity;
    float *p_best_pos, *p_best_fitness;
    hiprandState *states;


if (argc!=9)
    {
        printf("\n     Invalid number of arguments!");
    }

    N           = atoi(argv[1]);
    D           = atoi(argv[2]);
    xmax        = atoi(argv[3]);
    xmin        = atoi(argv[4]);
    c_1         = atoi(argv[5]);
    c_2         = atoi(argv[6]);
    inertia     = atoi(argv[7]);
    vmax        = atoi(argv[8]);

    // Calculated variables	
    float chi;
    chi = 2/abs(2-c_1 - c_2 - sqrt((c_2+c_1)*(c_2+c_1)-(4*c_2+4*c_1)));

//  MEMORY ALLOCATION

    // Allocating memory for all variables that do not come from input
    printf("Allocating memory for all other variables that do not come from input ..."); fflush(stdout);
    startTime(&timer); 

    //Particle position array (=Position in Git)
    hipMalloc((void**)&particle_position, N * D * sizeof(float));
    hipError_t err = hipGetLastError();        // Get error code
   if ( err != hipSuccess )
   {
      printf("CUDA Error in particle position array memory allocation: %s\n", hipGetErrorString(err));
      exit(-1);
   }

    //Velocity array (=Velocity in Git)
    hipMalloc((void**)&particle_velocity, N * D * sizeof(float));
    hipError_t err = hipGetLastError();        // Get error code
   if ( err != hipSuccess )
   {
      printf("CUDA Error in velocity array memory allocation: %s\n", hipGetErrorString(err));
      exit(-1);
   }
   
    //Particle best position array (=PBestX in git)
    hipMalloc((void**)&p_best_pos, N * D * sizeof(float));
    hipError_t err = hipGetLastError();        // Get error code
   if ( err != hipSuccess )
   {
      printf("CUDA Error in Particle best position array memory allocation: %s\n", hipGetErrorString(err));
      exit(-1);
   }
   
    //Particle best fitness value (=PBestY Array in Git)
    hipMalloc((void**)&p_best_fitness, N * sizeof(float));
    hipError_t err = hipGetLastError();        // Get error code
   if ( err != hipSuccess )
   {
      printf("CUDA Error in Particle best fitness value memory allocation: %s\n", hipGetErrorString(err));
      exit(-1);
   }

    //Local best index value (from a fintess point of view) (=LBestIndex in Git)
    hipMalloc((void**)&l_best_index, N * sizeof(int));
    hipError_t err = hipGetLastError();        // Get error code
   if ( err != hipSuccess )   
   {
      printf("CUDA Error in Local best index value memory allocation: %s\n", hipGetErrorString(err));
      exit(-1);
   }

    //Global Best index value (from a fintess point of view)(=GBestIndex in Git)
    hipMalloc((void**)&best_index, N * sizeof(int));
    hipError_t err = hipGetLastError();        // Get error code
   if ( err != hipSuccess )
   {
      printf("CUDA Error in Global best index value memory allocation: %s\n", hipGetErrorString(err));
      exit(-1);
   }
   
   // Cuda Rand memory allocation
    hipMalloc((void**)&states, N * D * sizeof(hiprandState));
    hipError_t err = hipGetLastError();        // Get error code
    if ( err != hipSuccess )
    {
      printf("Cuda Rand memory allocation: %s\n", hipGetErrorString(err));
      exit(-1);
    }   
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

//  Initialization of random numbers on GPU
    printf("Initialization of random numbers on GPU for particle's velocity and position ..."); fflush(stdout);
    startTime(&timer); 
    hiprandGenerator_t generator;
    hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(generator, time(NULL));

//  Initialize the particle's velocity (values of the array within the bounds): 
    hiprandGenerateUniform(generator, particle_velocity, N * D);
    hipError_t err = hipGetLastError();        // Get error code
   if ( err != hipSuccess )
   {
      printf("CUDA Error in random generation for intial particle velocity: %s\n", hipGetErrorString(err));
      exit(-1);
   }
//  Initialize the particle's position with a uniformly distributed random vector (values within the bounds)
    hiprandGenerateUniform(generator, particle_position, N * D);
    hipError_t err = hipGetLastError();        // Get error code
   if ( err != hipSuccess )
   {
      printf("CUDA Error in random generation for intial particle position: %s\n", hipGetErrorString(err));
      exit(-1);
   }
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));


// Scale velocity and position values to be between the max and min (and not between 0 and 1 anymore)
// AND initialize particle best (for each particle) + local best 
    printf("Launch kernel to scale and initialize ..."); fflush(stdout);
    startTime(&timer); 
    const unsigned int THREADS_PER_BLOCK = 200;
    const unsigned int numBlocks = N/THREADS_PER_BLOCK;
    dim3 gridDim(numBlocks, 1, 1), blockDim(THREADS_PER_BLOCK, 1, 1);    
    Scale_Init <<< gridDim, blockDim >>>(xmax, xmin, particle_position, particle_velocity, p_best_fitness, l_best_index, best_index, states);
    hipError_t err = hipGetLastError();        // Get error code
   if ( err != hipSuccess )
   {
      printf("CUDA Error Scale Init: %s\n", hipGetErrorString(err));
      exit(-1);
   }
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

//  Kernel for iterations
    printf("Launch kernel to compute iterations ..."); fflush(stdout);
    startTime(&timer); 
    for (int i = 0; i < max_iters; i++){
        Iterations<<< gridDim, blockDim >>>(xmax, xmin, particle_position, particle_velocity, p_best_pos, p_best_fitness, l_best_index, best_index, states, c_1, c_2, inertia, vmax, chi, N, D);
    }
   hipError_t err = hipGetLastError();        // Get error code
   if ( err != hipSuccess )
   {
      printf("CUDA Error in iterations: %s\n", hipGetErrorString(err));
      exit(-1);
   }
    hipDeviceSynchronize();
    
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("\nFreeing memory");

    //INSERT CODE HERE to free device matrices
    hipFree(particle_position);
    hipFree(particle_velocity);
    hipFree(p_best_pos);
    hipFree(p_best_fitness);
    hipFree(l_best_index);
    hipFree(best_index);
}
