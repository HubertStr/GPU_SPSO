#include "hip/hip_runtime.h"
// Let this script use GPU generated random numbers


#include <stdio.h>
#include "support.h"
#include "kernel.cu"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define max_iters 2048              //number of iterations

#define inf 9999.99f                //infinity


int main(int argc, char**argv){
    Timer timer;
    int N_h;
    int D_h;
    float xmax_h;
    float xmin_h;
    float c_1_h;
    float c_2_h;
    float inertia_h;int *local_best_index, *best_index;
    float *particle_position, *particle_velocity;
    float *p_best_pos, *p_best_fitness;
    hiprandState *states;


    if (argc<2)
    {
        printf("You need add 2 parameters\n");
        return 1;
    }
    for (int i=1; i<argc; i++){
        N_h= atoi(argv[i]);
        D_h= atoi(argv[++i]);
        xmax_h= atoi(argv[++i]);
        xmin_h = atoi(argv[++i]);
        c_1_h = atoi(argv[++i]);
        c_2_h = atoi(argv[++i]);
        inertia_h =  atoi(argv[++i]);
    }

//  MEMORY ALLOCATION

    printf("\nMEMORY ALLOCATION..."); fflush(stdout);
    startTime(&timer);

    // Host and device meory allocation of input variables
    printf("Device memory allocation of input variables..."); fflush(stdout);

    hipMalloc((void**)&N, sizeof(float));
    hipMalloc((void**)&D, sizeof(float));
    hipMalloc((void**)&xmax, sizeof(float));
    hipMalloc((void**)&xmin, sizeof(float));
    hipMalloc((void**)&c_1, sizeof(float));
    hipMalloc((void**)&c_2, sizeof(float));
    hipMalloc((void**)&inertia, sizeof(float));

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Migrate from host to device the inputs
    printf("Migrate from host to device the inputs..."); fflush(stdout);
    startTime(&timer);    

    cuda_ret = hipMemcpy(N, N_h, sizeof(float), hipMemcpyHostToDevice);
	if(cuda_ret != hipSuccess)
    {
      printf("CUDA Error in N memory allocation on device: %s\n", hipGetErrorString(err));
      exit(-1);
    }

    cuda_ret = hipMemcpy(D, D_h, sizeof(float), hipMemcpyHostToDevice);
	if(cuda_ret != hipSuccess)
    {
      printf("CUDA Error in D memory allocation on device: %s\n", hipGetErrorString(err));
      exit(-1);
    }

    cuda_ret = hipMemcpy(xmax, xmax_h, sizeof(float), hipMemcpyHostToDevice);
	if(cuda_ret != hipSuccess)
    {
      printf("CUDA Error in xmax memory allocation on device: %s\n", hipGetErrorString(err));
      exit(-1);
    }

    cuda_ret = hipMemcpy(xmin, xmin_h, sizeof(float), hipMemcpyHostToDevice);
	if(cuda_ret != hipSuccess)
    {
      printf("CUDA Error in xmin memory allocation on device: %s\n", hipGetErrorString(err));
      exit(-1);
    }

    cuda_ret = hipMemcpy(c_1, c_1_h, sizeof(float), hipMemcpyHostToDevice);
	if(cuda_ret != hipSuccess)
    {
      printf("CUDA Error in c_1 memory allocation on device: %s\n", hipGetErrorString(err));
      exit(-1);
    }

    cuda_ret = hipMemcpy(c_2, c_2_h, sizeof(float), hipMemcpyHostToDevice);
	if(cuda_ret != hipSuccess)
    {
      printf("CUDA Error in c_2 memory allocation on device: %s\n", hipGetErrorString(err));
      exit(-1);
    }

    cuda_ret = hipMemcpy(inertia, inertia_h, sizeof(float), hipMemcpyHostToDevice);
	if(cuda_ret != hipSuccess)
    {
      printf("CUDA Error in inertia memory allocation on device: %s\n", hipGetErrorString(err));
      exit(-1);
    }
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Allocating memory for all other variables that do not come from input
    printf("Allocating memory for all other variables that do not come from input ..."); fflush(stdout);
    startTime(&timer); 

    //Particle position array (=Position in Git)
    hipMalloc((void**)&particle_position, N * D * sizeof(float));
    hipError_t err = hipGetLastError();        // Get error code
   if ( err != hipSuccess )
   {
      printf("CUDA Error in particle position array memory allocation: %s\n", hipGetErrorString(err));
      exit(-1);
   }

    //Velocity array (=Velocity in Git)
    hipMalloc((void**)&particle_velocity, N * D * sizeof(float));
    hipError_t err = hipGetLastError();        // Get error code
   if ( err != hipSuccess )
   {
      printf("CUDA Error in velocity array memory allocation: %s\n", hipGetErrorString(err));
      exit(-1);
   }
   
    //Particle best position array (=PBestX in git)
    hipMalloc((void**)&p_best_pos, N * D * sizeof(float));
    hipError_t err = hipGetLastError();        // Get error code
   if ( err != hipSuccess )
   {
      printf("CUDA Error in Particle best position array memory allocation: %s\n", hipGetErrorString(err));
      exit(-1);
   }
   
    //Particle best fitness value (=PBestY Array in Git)
    hipMalloc((void**)&p_best_fitness, N * sizeof(float));
    hipError_t err = hipGetLastError();        // Get error code
   if ( err != hipSuccess )
   {
      printf("CUDA Error in Particle best fitness value memory allocation: %s\n", hipGetErrorString(err));
      exit(-1);
   }

    //Local best index value (from a fintess point of view) (=LBestIndex in Git)
    hipMalloc((void**)&l_best_index, N * sizeof(int));
    hipError_t err = hipGetLastError();        // Get error code
   if ( err != hipSuccess )   
   {
      printf("CUDA Error in Local best index value memory allocation: %s\n", hipGetErrorString(err));
      exit(-1);
   }

    //Global Best index value (from a fintess point of view)(=GBestIndex in Git)
    hipMalloc((void**)&best_index, N * sizeof(int));
    hipError_t err = hipGetLastError();        // Get error code
   if ( err != hipSuccess )
   {
      printf("CUDA Error in Global best index value memory allocation: %s\n", hipGetErrorString(err));
      exit(-1);
   }
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

//  Initialization of random numbers on GPU
    printf("Initialization of random numbers on GPU for particle's velocity and position ..."); fflush(stdout);
    startTime(&timer); 
    hiprandGenerator_t generator;
    hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(generator, time(NULL));

//  Initialize the particle's velocity (values of the array within the bounds): 
    hiprandGenerateUniform(generator, particle_velocity, N * D);
    hipError_t err = hipGetLastError();        // Get error code
   if ( err != hipSuccess )
   {
      printf("CUDA Error in random generation for intial particle velocity: %s\n", hipGetErrorString(err));
      exit(-1);
   }
//  Initialize the particle's position with a uniformly distributed random vector (values within the bounds)
    hiprandGenerateUniform(generator, particle_position, N * D);
    hipError_t err = hipGetLastError();        // Get error code
   if ( err != hipSuccess )
   {
      printf("CUDA Error in random generation for intial particle position: %s\n", hipGetErrorString(err));
      exit(-1);
   }
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));


// Scale velocity and position values to be between the max and min (and not between 0 and 1 anymore)
// AND initialize particle best (for each particle) + local best 
    printf("Launch kernel to scale and initialize ..."); fflush(stdout);
    startTime(&timer); 
    const unsigned int THREADS_PER_BLOCK = 200;
    const unsigned int numBlocks = N/THREADS_PER_BLOCK;
    dim3 gridDim(numBlocks, 1, 1), blockDim(THREADS_PER_BLOCK, 1, 1);    
    Scale_Init <<< gridDim, blockDim >>>(xmax, xmin, particle_position, particle_velocity, p_best_fitness, l_best_index, best_index, states);
    hipError_t err = hipGetLastError();        // Get error code
   if ( err != hipSuccess )
   {
      printf("CUDA Error Scale Init: %s\n", hipGetErrorString(err));
      exit(-1);
   }
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

//  Kernel for iterations
    printf("Launch kernel to compute iterations ..."); fflush(stdout);
    startTime(&timer); 
    for (int i = 0; i < max_iters; i++){
        Iterations<<< gridDim, blockDim >>>(xmax, xmin, particle_position, particle_velocity, p_best_pos, p_best_fitness, l_best_index, best_index, states, c_1, c_2);
    }
   hipError_t err = hipGetLastError();        // Get error code
   if ( err != hipSuccess )
   {
      printf("CUDA Error in iterations: %s\n", hipGetErrorString(err));
      exit(-1);
   }
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
}
